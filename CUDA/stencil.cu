#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include "kernels.h"
using namespace std;

#include <iostream>
using std::cout;
using std::endl;


#define GPU_RUN(call,benchmark_name, preproc, destroy) {\
    const int mem_size = len*sizeof(int); \
    int* arr_in  = (int*)malloc(mem_size*2); \
    int* arr_out = arr_in + len; /*(int*)malloc(mem_size);*/ \
    for(int i=0; i<len; i++){ arr_in[i] = i+1; } \
    int* gpu_array_in; \
    int* gpu_array_out; \
    CUDASSERT(hipMalloc((void **) &gpu_array_in, 2*mem_size)); \
    gpu_array_out = gpu_array_in + len; \
    /*CUDASSERT(hipMalloc((void **) &gpu_array_out, mem_size));*/ \
    CUDASSERT(hipMemcpy(gpu_array_in, arr_in, mem_size, hipMemcpyHostToDevice));\
    CUDASSERT(hipMemset(gpu_array_out, 0, mem_size));\
    (preproc);\
    CUDASSERT(hipDeviceSynchronize());\
    cout << (benchmark_name) << endl; \
    gettimeofday(&t_startpar, NULL); \
    for(unsigned x = 0; x < RUNS; x++){ \
        (call); \
    }\
    CUDASSERT(hipDeviceSynchronize());\
    gettimeofday(&t_endpar, NULL);\
    CUDASSERT(hipMemcpy(arr_out, gpu_array_out, mem_size, hipMemcpyDeviceToHost));\
    CUDASSERT(hipDeviceSynchronize());\
    timeval_subtract(&t_diffpar, &t_endpar, &t_startpar);\
    unsigned long elapsed = t_diffpar.tv_sec*1e6+t_diffpar.tv_usec;\
    elapsed /= RUNS;\
    printf("    mean elapsed time was: %lu microseconds\n", elapsed);\
    printf("%d %d %d %d %d %d\n", arr_out[0], arr_out[1], arr_out[2], arr_out[3],arr_out[10], arr_out[len-1]); \
    if (validate(cpu_out,arr_out,len)) \
    { \
        printf("%s\n", "VALIDATED");\
    }\
    free(arr_in);\
    /*free(arr_out);*/\
    hipFree(gpu_array_in);\
    /*hipFree(gpu_array_out);*/\
    (destroy);\
}


static int timeval_subtract(struct timeval *result, struct timeval *t2, struct timeval *t1)
{
    unsigned int resolution=1000000;
    long int diff = (t2->tv_usec + resolution * t2->tv_sec) - (t1->tv_usec + resolution * t1->tv_sec);
    result->tv_sec = diff / resolution;
    result->tv_usec = diff % resolution;
    return (diff<0);
}


static inline void cudAssert(hipError_t exit_code,
        const char *file,
        int         line) {
    if (exit_code != hipSuccess) {
        fprintf(stderr, ">>> Cuda run-time error: %s, at %s:%d\n",
                hipGetErrorString(exit_code), file, line);
        exit(exit_code);
    }
}
#define CUDASSERT(exit_code) { cudAssert((exit_code), __FILE__, __LINE__); }

bool validate(const int* A, const int* B, unsigned int sizeAB){
    int c = 0;
    for(unsigned i = 0; i < sizeAB; i++)
        if (A[i] != B[i]){
            printf("INVALID RESULT at index %d: (expected, actual) == (%d, %d)\n",
                    i, A[i], B[i]);
            c++;
            if(c > 20)
                return false;
        }
    return c == 0;
}

int stencil_fun_cpu(const int* arr, const int D)
{
    int sum_acc = 0;
    for (int i = 0; i < D; ++i){
        sum_acc += arr[i];
    }
    return sum_acc/(D);
}

template<int W>
void stencil_1d_cpu(
    const int* start,
    const int* idxs,
    int* out,
    const int len)
{
    int w2 = 2*W+1;
    for (int i = 0; i < len; ++i)
    {
        int arr[w2];
        for (int j = 0; j < w2; ++j)
        {
            int idx = idxs[j];
            int bound = min(len-1,max(0,i+idx));
            arr[j] = start[bound];
        }
        int lambda_res = stencil_fun_cpu(arr,w2);
        out[i] = lambda_res;
    }
}

template<int D, int block>
void stencil_1d_inSharedtiled(
    const int * start,
    const int * ixs,
    int * out,
    const unsigned len
    )
{
    const int working_block = block-(D-1);
    const int grid = (D + len + (working_block-1)) / working_block;

    inSharedtiled_1d<D,block><<<grid,block>>>(start, ixs, out, len);
    CUDASSERT(hipDeviceSynchronize());
}

#define call_kernel(kernel,blocksize) {\
    const int block = blocksize;\
    const int grid = (len + (block-1)) / block;\
    kernel;\
    CUDASSERT(hipDeviceSynchronize());\
}


template<int D, int block_size>
void stencil_1d_global_temp(
    const int * start,
    const int * ixs,
    int * temp,
    int * out,
    const int len
    )
{
    const int grid1 = (len*D + (block_size-1)) / block_size;
    const int grid2 = (len + (block_size-1)) / block_size;

    global_temp__1d_to_temp<D><<<grid1,block_size>>>(start, ixs, temp, len);
    global_temp__1d<D><<<grid2,block_size>>>(temp, out, len);
    CUDASSERT(hipDeviceSynchronize());
}

template<int W>
int* run_cpu(const int* idxs, const int len)
{
    int* cpu_in = (int*)malloc(len*sizeof(int));
    int* cpu_out = (int*)malloc(len*sizeof(int));

    for (int i = 0; i < len; ++i)
    {
        cpu_in[i] = i+1;
    }

    stencil_1d_cpu<W>(cpu_in,idxs,cpu_out,len);
    free(cpu_in);
    return cpu_out;
}


template<int W, int RUNS, int standard_block_size>
void doTest()
{
    struct timeval t_startpar, t_endpar, t_diffpar;

    const int D = (2*W+1);
    const int ixs_size = D*sizeof(int);
    int* ixs = (int*)malloc(ixs_size);
    for(int i=0; i < D ; i++){ ixs[i] = i-W; } \
    int* gpu_ixs;
    CUDASSERT(hipMalloc((void **) &gpu_ixs, ixs_size));
    CUDASSERT(hipMemcpy(gpu_ixs, ixs, ixs_size, hipMemcpyHostToDevice));

    const int len = 5000000;
    int* cpu_out = run_cpu<W>(ixs,len);

    {
//        GPU_RUN(call_kernel(
//                    (breathFirst_1d<W><<<grid,block>>>(gpu_array_in, gpu_ixs, gpu_array_out, len))
//                    ,standard_block_size)
//                ,"## Benchmark GPU 1d global-mem ##",(void)0,(void)0);
        GPU_RUN(call_kernel(
                    (big_tiled_1d<W,block><<<grid,block>>>(gpu_array_in, gpu_ixs, gpu_array_out, len))
                    ,standard_block_size)
                ,"## Benchmark GPU 1d big-tiled ##",(void)0,(void)0);
        GPU_RUN(call_kernel(
                    (inlinedIndexes_1d<W><<<grid,block>>>(gpu_array_in, gpu_ixs, gpu_array_out, len))
                    ,standard_block_size)
                ,"## Benchmark GPU 1d inlined idxs with global reads ##",(void)0,(void)0);
        GPU_RUN(call_kernel(
                    (threadLocalArr_1d<W><<<grid,block>>>(gpu_array_in, gpu_ixs, gpu_array_out, len))
                    ,standard_block_size)
                ,"## Benchmark GPU 1d local temp-array ##",(void)0,(void)0);
        GPU_RUN(call_kernel(
                    (outOfSharedtiled_1d<W,block><<<grid,block>>>(gpu_array_in, gpu_ixs, gpu_array_out, len))
                    ,standard_block_size)
                ,"## Benchmark GPU 1d out of shared tiled ##",(void)0,(void)0);
        GPU_RUN((stencil_1d_inSharedtiled<D, standard_block_size>(gpu_array_in, gpu_ixs, gpu_array_out, len)),
                "## Benchmark GPU 1d in shared tiled ##",(void)0,(void)0);
        /*GPU_RUN((stencil_1d_global_temp<D, standard_block_size>(gpu_array_in, gpu_ixs, temp, gpu_array_out, len)),
                "## Benchmark GPU 1d global temp ##"
                ,(CUDASSERT(hipMalloc((void **) &temp, D*mem_size)))
                ,(hipFree(temp)));*/
    }

    free(cpu_out);
    hipFree(gpu_ixs);
    free(ixs);
}

int main()
{
    doTest<5,1000,1024>();
    return 0;
}





/*static void sevenPointStencil(
        float * start,
        float * swap_out,
        const unsigned nx,
        const unsigned ny,
        const unsigned nz,
        const unsigned iterations // must be odd
        )
{
    const int T = 32;
    const int dimx = (nz + (T-1))/T;
    const int dimy = (ny + (T-1))/T;
    dim3 block(T,T,1);
    dim3 grid(dimx, dimy, 1);

    for (unsigned i = 0; i < iterations; ++i){
        if(i & 1){
            sevenPointStencil_single_iter<<< grid,block >>>(swap_out, start, nx, ny, nz);
        }
        else {
            sevenPointStencil_single_iter<<< grid,block >>>(start, swap_out, nx, ny, nz);
        }
    }
    CUDASSERT(hipDeviceSynchronize());

}

static void sevenPointStencil_tiledSliding(
        float * start,
        float * swap_out,
        const unsigned nx,
        const unsigned ny,
        const unsigned nz,
        const unsigned iterations // must be odd
        )
{
    const int T = 32;
    const int dimx = (nz + (T-1))/T;
    const int dimy = (ny + (T-1))/T;
    dim3 block(T,T,1);
    dim3 grid(dimx, dimy, 1);

    for (unsigned i = 0; i < iterations; ++i){
        if(i & 1){
            sevenPointStencil_single_iter_tiled_sliding <<< grid,block >>>(swap_out, start, nx, ny, nz);
        }
        else {
            sevenPointStencil_single_iter_tiled_sliding <<< grid,block >>>(start, swap_out, nx, ny, nz);
        }
    }
    CUDASSERT(hipDeviceSynchronize());

}
static void sevenPointStencil_tiledSliding_fully(
        float * start,
        float * swap_out,
        const unsigned nx,
        const unsigned ny,
        const unsigned nz,
        const unsigned iterations // must be odd
        )
{
    const unsigned T = 32;
    const unsigned Ts = 6;
    const unsigned dimx = (nx + (T-1))/T;
    const unsigned dimy = (ny + (Ts-1))/Ts;
    const unsigned dimz = (nz + (Ts-1))/Ts;
    dim3 block(32,6,6);
    dim3 grid(dimx, dimy, dimz);

    for (unsigned i = 0; i < iterations; ++i){
        if(i & 1){
            sevenPointStencil_single_iter_tiled_sliding_read<<<grid,block>>>(swap_out, start, nx, ny, nz);
        }
        else {
            sevenPointStencil_single_iter_tiled_sliding_read<<<grid,block>>>(start, swap_out, nx, ny, nz);
        }
    }
    CUDASSERT(hipDeviceSynchronize());

}*/
