#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include "kernels.h"
#include "kernels-2d.h"
using namespace std;

#include <iostream>
using std::cout;
using std::endl;


#define GPU_RUN(call,benchmark_name, preproc, destroy) {\
    const int mem_size = len*sizeof(int); \
    int* arr_in  = (int*)malloc(mem_size*2); \
    int* arr_out = arr_in + len; \
    for(int i=0; i<len; i++){ arr_in[i] = i+1; } \
    int* gpu_array_in; \
    int* gpu_array_out; \
    CUDASSERT(hipMalloc((void **) &gpu_array_in, 2*mem_size)); \
    gpu_array_out = gpu_array_in + len; \
    CUDASSERT(hipMemcpy(gpu_array_in, arr_in, mem_size, hipMemcpyHostToDevice));\
    CUDASSERT(hipMemset(gpu_array_out, 0, mem_size));\
    (preproc);\
    CUDASSERT(hipDeviceSynchronize());\
    cout << (benchmark_name) << endl; \
    gettimeofday(&t_startpar, NULL); \
    for(unsigned x = 0; x < RUNS; x++){ \
        (call); \
    }\
    CUDASSERT(hipDeviceSynchronize());\
    gettimeofday(&t_endpar, NULL);\
    CUDASSERT(hipMemcpy(arr_out, gpu_array_out, mem_size, hipMemcpyDeviceToHost));\
    CUDASSERT(hipDeviceSynchronize());\
    timeval_subtract(&t_diffpar, &t_endpar, &t_startpar);\
    unsigned long elapsed = t_diffpar.tv_sec*1e6+t_diffpar.tv_usec;\
    elapsed /= RUNS;\
    printf("    mean elapsed time was: %lu microseconds\n", elapsed);\
    printf("%d %d %d %d %d %d\n", arr_out[0], arr_out[1], arr_out[2], arr_out[3],arr_out[10], arr_out[len-1]); \
    if (validate(cpu_out,arr_out,len)) \
    { \
        printf("%s\n", "VALIDATED");\
    }\
    free(arr_in);\
    CUDASSERT(hipFree(gpu_array_in));\
    (destroy);\
}

#define GPU_RUN_2D(call,benchmark_name) {\
    const int mem_size = len*sizeof(int); \
    int* arr_in  = (int*)malloc(mem_size*2); \
    int* arr_out = arr_in + len; \
    for(int i=0; i<len; i++){ arr_in[i] = i+1; } \
    int* gpu_array_in; \
    int* gpu_array_out; \
    CUDASSERT(hipMalloc((void **) &gpu_array_in, 2*mem_size)); \
    gpu_array_out = gpu_array_in + len; \
    CUDASSERT(hipMemcpy(gpu_array_in, arr_in, mem_size, hipMemcpyHostToDevice));\
    CUDASSERT(hipMemset(gpu_array_out, 0, mem_size));\
    (preproc);\
    CUDASSERT(hipDeviceSynchronize());\
    cout << (benchmark_name) << endl; \
    gettimeofday(&t_startpar, NULL); \
    for(unsigned x = 0; x < RUNS; x++){ \
        (call); \
    }\
    CUDASSERT(hipDeviceSynchronize());\
    gettimeofday(&t_endpar, NULL);\
    CUDASSERT(hipMemcpy(arr_out, gpu_array_out, mem_size, hipMemcpyDeviceToHost));\
    CUDASSERT(hipDeviceSynchronize());\
    timeval_subtract(&t_diffpar, &t_endpar, &t_startpar);\
    unsigned long elapsed = t_diffpar.tv_sec*1e6+t_diffpar.tv_usec;\
    elapsed /= RUNS;\
    printf("    mean elapsed time was: %lu microseconds\n", elapsed);\
    printf("%d %d %d %d %d %d\n", arr_out[0], arr_out[1], arr_out[2], arr_out[3],arr_out[10], arr_out[len-1]); \
    if (validate(cpu_out,arr_out,len)) \
    { \
        printf("%s\n", "VALIDATED");\
    }\
    free(arr_in);\
    CUDASSERT(hipFree(gpu_array_in));\
    (destroy);\
}


static int timeval_subtract(struct timeval *result, struct timeval *t2, struct timeval *t1)
{
    unsigned int resolution=1000000;
    long int diff = (t2->tv_usec + resolution * t2->tv_sec) - (t1->tv_usec + resolution * t1->tv_sec);
    result->tv_sec = diff / resolution;
    result->tv_usec = diff % resolution;
    return (diff<0);
}


static inline void cudAssert(hipError_t exit_code,
        const char *file,
        int         line) {
    if (exit_code != hipSuccess) {
        fprintf(stderr, ">>> Cuda run-time error: %s, at %s:%d\n",
                hipGetErrorString(exit_code), file, line);
        exit(exit_code);
    }
}
#define CUDASSERT(exit_code) { cudAssert((exit_code), __FILE__, __LINE__); }

bool validate(const int* A, const int* B, unsigned int sizeAB){
    int c = 0;
    for(unsigned i = 0; i < sizeAB; i++)
        if (A[i] != B[i]){
            printf("INVALID RESULT at index %d: (expected, actual) == (%d, %d)\n",
                    i, A[i], B[i]);
            c++;
            if(c > 20)
                return false;
        }
    return c == 0;
}

int stencil_fun_cpu(const int* arr, const int D)
{
    int sum_acc = 0;
    for (int i = 0; i < D; ++i){
        sum_acc += arr[i];
    }
    return sum_acc/(D);
}

template<int D>
void stencil_1d_cpu(
    const int* start,
    const int* idxs,
    int* out,
    const int len)
{
    for (int i = 0; i < len; ++i)
    {
        int arr[D];
        for (int j = 0; j < D; ++j)
        {
            int idx = idxs[j];
            int bound = min(len-1,max(0,i+idx));
            arr[j] = start[bound];
        }
        int lambda_res = stencil_fun_cpu(arr,D);
        out[i] = lambda_res;
    }
}

template<int W>
void stencil_2d_cpu(
    const int* start,
    const int* idxs,
    int* out,
    const int n_rows,
    const int n_columns)
{
    int w2 = 2*W+1;
    for (int i = 0; i < n_rows; ++i)
    {
        for (int k = 0; k < n_columns; ++k)
        {
            int arr[w2];
            for (int j = 0; j < w2; ++j)
            {
                int idx = idxs[j];
                int bound = min(n_rows*n_columns - 1,max(0,i*n_columns + k + idx));
                arr[j] = start[bound];
            }
            int lambda_res = stencil_fun_cpu(arr,w2);
            out[i] = lambda_res;
        }
    }
}

template<int ixs_len, int ix_min, int ix_max>
void stencil_1d_inSharedtiled(
    const int * start,
    const int * ixs,
    int * out,
    const unsigned len
    )
{
    const int wasted = ix_min + ix_max;
    const int working_block = T-wasted;
    const int grid = (ixs_len + len + (working_block-1)) / working_block;

    inSharedtiled_1d<ixs_len,ix_min,ix_max><<<grid,T>>>(start, ixs, out, len);
    CUDASSERT(hipDeviceSynchronize());
}

template<int ixs_len, int ix_min, int ix_max>
void stencil_1d_inSharedtiled_const_ixs_inline(
    const int * start,
    int * out,
    const unsigned len
    )
{
    const int wasted = ix_min + ix_max;
    const int working_block = T-wasted;
    const int grid = (ixs_len + len + (working_block-1)) / working_block;

    inSharedtiled_1d_const_ixs_inline<ixs_len,ix_min,ix_max><<<grid,T>>>(start, out, len);
    CUDASSERT(hipDeviceSynchronize());
}

template<int ixs_len, int ix_min, int ix_max>
void stencil_1d_inSharedtiled_const_ixs(
    const int * start,
    int * out,
    const unsigned len
    )
{
    const int wasted = ix_min + ix_max;
    const int working_block = T-wasted;
    const int grid = (ixs_len + len + (working_block-1)) / working_block;

    inSharedtiled_1d_const_ixs<ixs_len,ix_min,ix_max><<<grid,T>>>(start, out, len);
    CUDASSERT(hipDeviceSynchronize());
}

#define call_kernel(kernel,blocksize) {\
    const int block = blocksize;\
    const int grid = (len + (block-1)) / block;\
    kernel;\
    CUDASSERT(hipDeviceSynchronize());\
}


template<int D>
void stencil_1d_global_temp(
    const int * start,
    const int * ixs,
    int * temp,
    int * out,
    const int len
    )
{
    const int grid1 = (len*D + (T-1)) / T;
    const int grid2 = (len + (T-1)) / T;

    global_temp__1d_to_temp<D><<<grid1,T>>>(start, ixs, temp, len);
    global_temp__1d<D><<<grid2,T>>>(temp, out, len);
    CUDASSERT(hipDeviceSynchronize());
}

template<int W>
int* run_cpu(const int* idxs, const int len)
{
    int* cpu_in = (int*)malloc(len*sizeof(int));
    int* cpu_out = (int*)malloc(len*sizeof(int));

    for (int i = 0; i < len; ++i)
    {
        cpu_in[i] = i+1;
    }

    stencil_1d_cpu<W>(cpu_in,idxs,cpu_out,len);
    free(cpu_in);
    return cpu_out;
}

template<int W>
int* run_cpu_2d(const int* idxs, const int n_rows, const int n_columns)
{
    int len = n_rows*n_columns;
    int* cpu_in = (int*)malloc(len*sizeof(int));
    int* cpu_out = (int*)malloc(len*sizeof(int));

    for (int i = 0; i < len; ++i)
    {
        cpu_in[i] = i+1;
    }

    stencil_2d_cpu<W>(cpu_in,idxs,cpu_out,n_rows,n_columns);
    free(cpu_in);
    return cpu_out;
}


template<int ixs_len, int ix_min, int ix_max>
void doAllTest()
{
    const int RUNS = 100;
    const int standard_block_size = T;

    struct timeval t_startpar, t_endpar, t_diffpar;

    const int D = ixs_len;
    const int ixs_size = D*sizeof(int);
    int* cpu_ixs = (int*)malloc(ixs_size);
    for(int i=0; i < D ; i++){ cpu_ixs[i] = i; }

    for(int i=0; i < D ; i++){
        const int V = cpu_ixs[i];
        if(-ix_min <= V && V <= ix_max)
        {}
        else { printf("index array contains indexes not in range\n"); }
    }
    int* gpu_ixs;
    CUDASSERT(hipMalloc((void **) &gpu_ixs, ixs_size));
    CUDASSERT(hipMemcpy(gpu_ixs, cpu_ixs, ixs_size, hipMemcpyHostToDevice));
    CUDASSERT(hipMemcpyToSymbol(HIP_SYMBOL(ixs), cpu_ixs, ixs_size));

    const int len = 5000000;
    int* cpu_out = run_cpu<D>(cpu_ixs,len);
    printf("%d %d %d %d %d %d\n", cpu_out[0], cpu_out[1], cpu_out[2], cpu_out[3],cpu_out[10], cpu_out[len-1]);

    cout << "D=" << D << endl;
    cout << "W=" << (D/2) << endl;
    {
        GPU_RUN(call_kernel(
                    (big_tiled_1d<ixs_len,ix_min,ix_max><<<grid,block>>>(gpu_array_in, gpu_ixs, gpu_array_out, len))
                    ,standard_block_size)
                ,"## Benchmark GPU 1d big-tiled ##",(void)0,(void)0);
        GPU_RUN(call_kernel(
                    (big_tiled_1d_const_ixs<ixs_len,ix_min,ix_max><<<grid,block>>>(gpu_array_in, gpu_array_out, len))
                    ,standard_block_size)
                ,"## Benchmark GPU 1d big-tiled const ixs ##",(void)0,(void)0);
        GPU_RUN(call_kernel(
                    (big_tiled_1d_const_ixs<ixs_len,ix_min,ix_max><<<grid,block>>>(gpu_array_in, gpu_array_out, len))
                    ,standard_block_size)
                ,"## Benchmark GPU 1d big-tiled const inline ixs ##",(void)0,(void)0);
        GPU_RUN(call_kernel(
                    (inlinedIndexes_1d<ixs_len><<<grid,block>>>(gpu_array_in, gpu_ixs, gpu_array_out, len))
                    ,standard_block_size)
                ,"## Benchmark GPU 1d inlined idxs with global reads ##",(void)0,(void)0);
        GPU_RUN(call_kernel(
                    (inlinedIndexes_1d_const_ixs<ixs_len><<<grid,block>>>(gpu_array_in, gpu_array_out, len))
                    ,standard_block_size)
                ,"## Benchmark GPU 1d inlined idxs with global reads const ixs ##",(void)0,(void)0);
        GPU_RUN(call_kernel(
                    (threadLocalArr_1d<ixs_len><<<grid,block>>>(gpu_array_in, gpu_ixs, gpu_array_out, len))
                    ,standard_block_size)
                ,"## Benchmark GPU 1d local temp-array w/ global reads ##",(void)0,(void)0);
        GPU_RUN(call_kernel(
                    (threadLocalArr_1d_const_ixs<ixs_len><<<grid,block>>>(gpu_array_in, gpu_array_out, len))
                    ,standard_block_size)
                ,"## Benchmark GPU 1d local temp-array const ixs w/ global reads ##",(void)0,(void)0);
        GPU_RUN(call_kernel(
                    (outOfSharedtiled_1d<ixs_len><<<grid,block>>>(gpu_array_in, gpu_ixs, gpu_array_out, len))
                    ,standard_block_size)
                ,"## Benchmark GPU 1d out of shared tiled /w local temp-array ##",(void)0,(void)0);
        GPU_RUN(call_kernel(
                    (outOfSharedtiled_1d_const_ixs<ixs_len><<<grid,block>>>(gpu_array_in, gpu_array_out, len))
                    ,standard_block_size)
                ,"## Benchmark GPU 1d out of shared tiled const ixs /w local temp-array ##",(void)0,(void)0);
        GPU_RUN((stencil_1d_inSharedtiled<ixs_len,ix_min,ix_max>(gpu_array_in, gpu_ixs, gpu_array_out, len)),
                "## Benchmark GPU 1d in shared tiled /w local temp-array ##",(void)0,(void)0);
        GPU_RUN((stencil_1d_inSharedtiled_const_ixs<ixs_len,ix_min,ix_max>(gpu_array_in, gpu_array_out, len)),
                "## Benchmark GPU 1d in shared tiled const ixs /w local temp-array ##",(void)0,(void)0);
        GPU_RUN((stencil_1d_inSharedtiled_const_ixs_inline<ixs_len,ix_min,ix_max>(gpu_array_in, gpu_array_out, len)),
                "## Benchmark GPU 1d in shared tiled const inline ixs ##",(void)0,(void)0);
        /*GPU_RUN((stencil_1d_global_temp<D, standard_block_size>(gpu_array_in, gpu_ixs, temp, gpu_array_out, len)),
                "## Benchmark GPU 1d global temp ##"
                ,(CUDASSERT(hipMalloc((void **) &temp, D*mem_size)))
                ,(hipFree(temp)));*/
    }

    free(cpu_out);
    hipFree(gpu_ixs);
    free(cpu_ixs);
}

template<int ixs_len, int ix_min, int ix_max>
void doTest()
{
    const int RUNS = 100;
    const int standard_block_size = T;

    struct timeval t_startpar, t_endpar, t_diffpar;

    const int D = ixs_len;
    const int ixs_size = D*sizeof(int);
    int* cpu_ixs = (int*)malloc(ixs_size);
    for(int i=0; i < D ; i++){ cpu_ixs[i] = i; }

    for(int i=0; i < D ; i++){
        const int V = cpu_ixs[i];
        if(-ix_min <= V && V <= ix_max)
        {}
        else { printf("index array contains indexes not in range\n"); }
    }
    CUDASSERT(hipMemcpyToSymbol(HIP_SYMBOL(ixs), cpu_ixs, ixs_size));

    const int len = 5000000;
    int* cpu_out = run_cpu<D>(cpu_ixs,len);
    printf("%d %d %d %d %d %d\n", cpu_out[0], cpu_out[1], cpu_out[2], cpu_out[3],cpu_out[10], cpu_out[len-1]);

    cout << "D=" << D << endl;
    cout << "W=" << (D/2) << endl;
    {
        GPU_RUN(call_kernel(
                    (inlinedIndexes_1d_const_ixs<ixs_len><<<grid,block>>>(gpu_array_in, gpu_array_out, len))
                    ,standard_block_size)
                ,"## Benchmark GPU 1d inlined idxs with global reads const ixs ##",(void)0,(void)0);
        GPU_RUN((stencil_1d_inSharedtiled_const_ixs_inline<ixs_len,ix_min,ix_max>(gpu_array_in, gpu_array_out, len)),
                "## Benchmark GPU 1d in shared tiled const inline ixs ##",(void)0,(void)0);
    }

    free(cpu_out);
    free(cpu_ixs);
}

template<int ixs_len, int ix_min, int ix_max>
void doTest_2D()
{
    const int RUNS = 100;
    const int standard_block_size = 1024;

    struct timeval t_startpar, t_endpar, t_diffpar;

    const int D = ixs_len;
    const int W = D / 2;
    const int ixs_size = D*sizeof(int);
    int* cpu_ixs = (int*)malloc(ixs_size);
    for(int i=0; i < D ; i++){ cpu_ixs[i] = i-W; } \
    int* gpu_ixs;
    CUDASSERT(hipMalloc((void **) &gpu_ixs, ixs_size));
    CUDASSERT(hipMemcpy(gpu_ixs, cpu_ixs, ixs_size, hipMemcpyHostToDevice));
    CUDASSERT(hipMemcpyToSymbol(HIP_SYMBOL(ixs), cpu_ixs, ixs_size));

    const int n_rows = 1000;
    const int n_columns = 1000;
    int* cpu_out = run_cpu_2d<W>(cpu_ixs,n_rows,n_columns);

    cout << "D=" << D << endl;
    cout << "W=" << W << endl;

}

int main()
{
    //doAllTest<4,5,5>();
    doTest<4,5,5>();
    return 0;
}





/*static void sevenPointStencil(
        float * start,
        float * swap_out,
        const unsigned nx,
        const unsigned ny,
        const unsigned nz,
        const unsigned iterations // must be odd
        )
{
    const int T = 32;
    const int dimx = (nz + (T-1))/T;
    const int dimy = (ny + (T-1))/T;
    dim3 block(T,T,1);
    dim3 grid(dimx, dimy, 1);

    for (unsigned i = 0; i < iterations; ++i){
        if(i & 1){
            sevenPointStencil_single_iter<<< grid,block >>>(swap_out, start, nx, ny, nz);
        }
        else {
            sevenPointStencil_single_iter<<< grid,block >>>(start, swap_out, nx, ny, nz);
        }
    }
    CUDASSERT(hipDeviceSynchronize());

}

static void sevenPointStencil_tiledSliding(
        float * start,
        float * swap_out,
        const unsigned nx,
        const unsigned ny,
        const unsigned nz,
        const unsigned iterations // must be odd
        )
{
    const int T = 32;
    const int dimx = (nz + (T-1))/T;
    const int dimy = (ny + (T-1))/T;
    dim3 block(T,T,1);
    dim3 grid(dimx, dimy, 1);

    for (unsigned i = 0; i < iterations; ++i){
        if(i & 1){
            sevenPointStencil_single_iter_tiled_sliding <<< grid,block >>>(swap_out, start, nx, ny, nz);
        }
        else {
            sevenPointStencil_single_iter_tiled_sliding <<< grid,block >>>(start, swap_out, nx, ny, nz);
        }
    }
    CUDASSERT(hipDeviceSynchronize());

}
static void sevenPointStencil_tiledSliding_fully(
        float * start,
        float * swap_out,
        const unsigned nx,
        const unsigned ny,
        const unsigned nz,
        const unsigned iterations // must be odd
        )
{
    const unsigned T = 32;
    const unsigned Ts = 6;
    const unsigned dimx = (nx + (T-1))/T;
    const unsigned dimy = (ny + (Ts-1))/Ts;
    const unsigned dimz = (nz + (Ts-1))/Ts;
    dim3 block(32,6,6);
    dim3 grid(dimx, dimy, dimz);

    for (unsigned i = 0; i < iterations; ++i){
        if(i & 1){
            sevenPointStencil_single_iter_tiled_sliding_read<<<grid,block>>>(swap_out, start, nx, ny, nz);
        }
        else {
            sevenPointStencil_single_iter_tiled_sliding_read<<<grid,block>>>(start, swap_out, nx, ny, nz);
        }
    }
    CUDASSERT(hipDeviceSynchronize());

}*/
