#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include "runners.h"
#include "kernels-2d.h"
using namespace std;
#include <iostream>
using std::cout;
using std::endl;


template<int D>
void stencil_2d_cpu(
    const T* start,
    const int2* idxs,
    T* out,
    const int y_len,
    const int x_len)
{
    const int max_y_ix = y_len - 1;
    const int max_x_ix = x_len - 1;
    for (int i = 0; i < y_len; ++i)
    {
        for (int k = 0; k < x_len; ++k)
        {
            T arr[D];
            for (int j = 0; j < D; ++j)
            {
                int y = BOUND(i + idxs[j].y, max_y_ix);
                int x = BOUND(k + idxs[j].x, max_x_ix);
                int index = y * x_len + x;
                arr[j] = start[index];
            }
            T lambda_res = stencil_fun_cpu<D>(arr);
            out[i * x_len + k] = lambda_res;
        }
    }
}

#define call_kernel_2d(kernel) {\
    const dim3 block(SQ_BLOCKSIZE,SQ_BLOCKSIZE,1);\
    const int BNx = CEIL_DIV(x_len, SQ_BLOCKSIZE);\
    const int BNy = CEIL_DIV(y_len, SQ_BLOCKSIZE);\
    const dim3 grid(BNx, BNy, 1);\
    kernel;\
    CUDASSERT(hipDeviceSynchronize());\
}
#define call_small_tile_2d(kernel) {\
    const dim3 block(SQ_BLOCKSIZE,SQ_BLOCKSIZE,1);\
    const int wasted_x = x_min + x_max;\
    const int wasted_y = y_min + y_max;\
    const int working_block_x = SQ_BLOCKSIZE-wasted_x;\
    const int working_block_y = SQ_BLOCKSIZE-wasted_y;\
    const int BNx = CEIL_DIV(x_len, working_block_x);\
    const int BNy = CEIL_DIV(y_len   , working_block_y);\
    const dim3 grid(BNx, BNy, 1);\
    kernel;\
    CUDASSERT(hipDeviceSynchronize());\
}

template<int D>
void run_cpu_2d(const int2* idxs, const int y_len, const int x_len, T* cpu_out)
{
    int len = y_len*x_len;
    T* cpu_in = (T*)malloc(len*sizeof(T));

    for (int i = 0; i < len; ++i)
    {
        cpu_in[i] = (T)(i+1);
    }

    struct timeval t_startpar, t_endpar, t_diffpar;
    gettimeofday(&t_startpar, NULL);
    {
        stencil_2d_cpu<D>(cpu_in,idxs,cpu_out,y_len,x_len);
    }
    gettimeofday(&t_endpar, NULL);
    timeval_subtract(&t_diffpar, &t_endpar, &t_startpar);
    const unsigned long elapsed = (t_diffpar.tv_sec*1e6+t_diffpar.tv_usec) / 1000;
    const unsigned long seconds = elapsed / 1000;
    const unsigned long microseconds = elapsed % 1000;
    printf("cpu c 2d for 1 run : %lu.%03lu seconds\n", seconds, microseconds);

    free(cpu_in);
}

template<int y_min, int y_max, int x_min, int x_max>
void doTest_2D()
{
    const int RUNS = 100;

    const int y_range = (y_min + y_max + 1);
    const int x_range = (x_min + x_max + 1);
    const int ixs_len = y_range * x_range;
    //const int W = D / 2;
    const int ixs_size = ixs_len*sizeof(int2);
    int2* cpu_ixs = (int2*)malloc(ixs_size);
    {
        int q = 0;
        for(int i=0; i < y_range; i++){
            for(int j=0; j < x_range; j++){
                cpu_ixs[q++] = make_int2(j-x_min, i-y_min);
            }
        }
    }

    CUDASSERT(hipMemcpyToSymbol(HIP_SYMBOL(ixs_2d), cpu_ixs, ixs_size));

    cout << "const int ixs[" << ixs_len << "]: ";
    cout << "y= " << -y_min << "..." << y_max << ", x= " << -x_min << "..." << x_max << endl;

    const int y_len = 1 << 14;
    const int x_len = 1 << 10;
    const int len = y_len * x_len;
    cout << "{ x_len = " << x_len << ", y_len = " << y_len
         << ", total_len = " << len << " }" << endl;
    T* cpu_out = (T*)malloc(len*sizeof(T));
    run_cpu_2d<ixs_len>(cpu_ixs,y_len,x_len, cpu_out);

    {
        GPU_RUN_INIT;
        /*
        GPU_RUN(call_kernel_2d(
                    (global_reads_2d<ixs_len><<<grid,block>>>(gpu_array_in, gpu_array_out, x_len, y_len)))
                ,"## Benchmark 2d global read ##",(void)0,(void)0);
        GPU_RUN(call_small_tile_2d(
                    (small_tile_2d<ixs_len,x_min,x_max,y_min,y_max><<<grid,block>>>(gpu_array_in, gpu_array_out, x_len, y_len)))
                ,"## Benchmark 2d small tile ##",(void)0,(void)0);
        GPU_RUN(call_kernel_2d(
                    (big_tile_2d<ixs_len,x_min,x_max,y_min,y_max><<<grid,block>>>(gpu_array_in, gpu_array_out, x_len, y_len)))
                ,"## Benchmark 2d big tile ##",(void)0,(void)0);
        */
        GPU_RUN(call_kernel_2d(
                    (global_reads_2d_const<x_min,x_max,y_min,y_max><<<grid,block>>>(gpu_array_in, gpu_array_out, x_len, y_len)))
                ,"## Benchmark 2d global read constant ixs ##",(void)0,(void)0);
        GPU_RUN(call_small_tile_2d(
                    (small_tile_2d_const<x_min,x_max,y_min,y_max><<<grid,block>>>(gpu_array_in, gpu_array_out, x_len, y_len)))
                ,"## Benchmark 2d small tile constant ixs ##",(void)0,(void)0);
        GPU_RUN(call_kernel_2d(
                    (big_tile_2d_const<x_min,x_max,y_min,y_max><<<grid,block>>>(gpu_array_in, gpu_array_out, x_len, y_len)))
                ,"## Benchmark 2d big tile constant ixs ##",(void)0,(void)0);
        GPU_RUN_END;
    }

    free(cpu_out);
    free(cpu_ixs);
}


int main()
{
    doTest_2D<1,1,0,0>();
    doTest_2D<2,2,0,0>();
    doTest_2D<3,3,0,0>();
    doTest_2D<4,4,0,0>();
    doTest_2D<5,5,0,0>();

    doTest_2D<1,1,1,1>();
    doTest_2D<2,2,2,2>();
    doTest_2D<3,3,3,3>();
    doTest_2D<4,4,4,4>();
    doTest_2D<5,5,5,5>();

    return 0;
}


