#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>

#include "runners.h"
#include "kernels-1d.h"

using namespace std;
#include <iostream>
using std::cout;
using std::endl;


template<int D>
void stencil_1d_cpu(
    const T* start,
    const int* idxs,
    T* out,
    const int len)
{
    for (int i = 0; i < len; ++i)
    {
        T arr[D];
        for (int j = 0; j < D; ++j)
        {
            int idx = idxs[j];
            int bound = min(len-1,max(0,i+idx));
            arr[j] = start[bound];
        }
        T lambda_res = stencil_fun_cpu<D>(arr);
        out[i] = lambda_res;
    }
}

#define call_inSharedKernel_1d(kernel) {\
    const int block = BLOCKSIZE;\
    const int wasted = ix_min + ix_max;\
    const int working_block = BLOCKSIZE-wasted;\
    const int grid = (wasted + len + (working_block-1)) / working_block;\
    kernel;\
    CUDASSERT(hipDeviceSynchronize());\
}

#define call_kernel_1d(kernel) {\
    const int block = BLOCKSIZE;\
    const int grid = (len + (block-1)) / block;\
    kernel;\
    CUDASSERT(hipDeviceSynchronize());\
}

template<int D>
T* run_cpu_1d(const int* idxs, const int len)
{
    T* cpu_in  = (T*)malloc(len*sizeof(T));
    T* cpu_out = (T*)malloc(len*sizeof(T));

    for (int i = 0; i < len; ++i)
    {
        cpu_in[i] = (T)(i+1);
    }

    stencil_1d_cpu<D>(cpu_in,idxs,cpu_out,len);
    free(cpu_in);
    return cpu_out;
}

template<int ixs_len, int ix_min, int ix_max>
void doTest_1D()
{
    const int RUNS = 100;

    const int D = ixs_len;
    const int ixs_size = D*sizeof(int);
    int* cpu_ixs = (int*)malloc(ixs_size);
    const int step = (ix_min + ix_max) / (ixs_len-1);
    {
        int s = -ix_min;
        for(int i=0; i < D ; i++){ cpu_ixs[i] = s; s += step; }
    }
    for(int i=0; i < D ; i++){
        const int V = cpu_ixs[i];
        if(-ix_min <= V && V <= ix_max)
        {}
        else { printf("index array contains indexes not in range\n"); exit(1);}
    }
    CUDASSERT(hipMemcpyToSymbol(HIP_SYMBOL(ixs_1d), cpu_ixs, ixs_size));

    const int lenp = 22;
    const int len = 2 << lenp;
    T* cpu_out = run_cpu_1d<D>(cpu_ixs,len);

    cout << "input[2^" << lenp << "]" << endl;
    //cout << "ixs[" << D << "]" << endl;
    cout << "ixs[" << D << "] = [";
    for(int i=0; i < D ; i++){
        cout << " " << cpu_ixs[i];
        if(i == D-1)
        { cout << "]" << endl; }
        else{ cout << ", "; }
    }

    {
        GPU_RUN_INIT;

        GPU_RUN(call_kernel_1d(
                    (inlinedIndexes_1d_const_ixs<ixs_len><<<grid,block>>>(gpu_array_in, gpu_array_out, len)))
                ,"## Benchmark 1d global reads ##",(void)0,(void)0);
        GPU_RUN(call_inSharedKernel_1d(
                    (inSharedtiled_1d_const_ixs_inline<ixs_len,ix_min,ix_max><<<grid,block>>>(gpu_array_in, gpu_array_out, len)))
                ,"## Benchmark 1d small tile ##",(void)0,(void)0);
        GPU_RUN(call_kernel_1d(
                    (big_tiled_1d_const_ixs_inline<ixs_len,ix_min,ix_max><<<grid,block>>>(gpu_array_in, gpu_array_out, len)))
                ,"## Benchmark 1d big tile ##",(void)0,(void)0);

        if(ixs_len == ix_min + ix_max + 1){
            GPU_RUN(call_kernel_1d(
                        (global_read_1d_const<ixs_len,ix_min,ix_max><<<grid,block>>>(gpu_array_in, gpu_array_out, len)))
                    ,"## Benchmark 1d global reads constant ixs ##",(void)0,(void)0);
            GPU_RUN(call_inSharedKernel_1d(
                        (small_tile_1d_const<ixs_len,ix_min,ix_max><<<grid,block>>>(gpu_array_in, gpu_array_out, len)))
                    ,"## Benchmark 1d small tile constant ixs  ##",(void)0,(void)0);
            GPU_RUN(call_kernel_1d(
                        (big_tile_1d_const<ixs_len,ix_min,ix_max><<<grid,block>>>(gpu_array_in, gpu_array_out, len)))
                    ,"## Benchmark 1d big tile constant ixs ##",(void)0,(void)0);
        }

        GPU_RUN_END;
    }

    free(cpu_out);
    free(cpu_ixs);
}


int main()
{
    doTest_1D<3,1,1>();
    doTest_1D<5,2,2>();
    doTest_1D<7,3,3>();
    doTest_1D<9,4,4>();
    doTest_1D<21,10,10>();
    doTest_1D<23,11,11>();
    doTest_1D<25,12,12>();
    doTest_1D<27,13,13>();
    doTest_1D<29,14,14>();
    doTest_1D<31,15,15>();
    doTest_1D<41,20,20>();
    doTest_1D<3,256,256>();

    return 0;
}

